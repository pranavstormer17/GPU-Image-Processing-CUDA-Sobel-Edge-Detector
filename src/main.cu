#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 * Command-line front-end:
 *  --input <path>
 *  --output <path>
 *  --block <n> (16 default)
 *  --log <path> (results/timings.csv default)
 */

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <string>
#include <fstream>
#include <iostream>
#include "sobel.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

static void print_usage(const char* prog) {
  std::printf("Usage: %s --input <infile> --output <outfile> [--block <n>] [--log <log.csv>]\n", prog);
}

int main(int argc, char** argv) {
  if (argc < 5) {
    print_usage(argv[0]);
    return 1;
  }

  std::string input_path, output_path, log_path = "results/timings.csv";
  int block_size = 16;

  for (int i = 1; i < argc; ++i) {
    if (std::strcmp(argv[i], "--input") == 0 && i + 1 < argc) {
      input_path = argv[++i];
    } else if (std::strcmp(argv[i], "--output") == 0 && i + 1 < argc) {
      output_path = argv[++i];
    } else if (std::strcmp(argv[i], "--block") == 0 && i + 1 < argc) {
      block_size = std::atoi(argv[++i]);
    } else if (std::strcmp(argv[i], "--log") == 0 && i + 1 < argc) {
      log_path = argv[++i];
    } else {
      print_usage(argv[0]);
      return 1;
    }
  }

  if (input_path.empty() || output_path.empty()) {
    print_usage(argv[0]);
    return 1;
  }

  int w, h, comp;
  unsigned char* img = stbi_load(input_path.c_str(), &w, &h, &comp, 4);
  if (!img) {
    std::cerr << "Failed to load image: " << input_path << std::endl;
    return 1;
  }
  std::cout << "Loaded " << input_path << " (" << w << " x " << h << "), channels forced to RGBA.\n";

  // output buffer
  size_t out_bytes = static_cast<size_t>(w) * h * 4;
  uint8_t* out_buf = (uint8_t*)malloc(out_bytes);
  if (!out_buf) {
    std::cerr << "Failed to allocate output buffer\n";
    stbi_image_free(img);
    return 1;
  }

  float gpu_ms = 0.0f;
  sobel_process_image(img, out_buf, w, h, w * 4, w * 4, block_size, &gpu_ms);

  // write output image
  // use PNG
  int write_rc = stbi_write_png(output_path.c_str(), w, h, 4, out_buf, w * 4);
  if (!write_rc) {
    std::cerr << "Failed to write output image: " << output_path << std::endl;
    free(out_buf);
    stbi_image_free(img);
    return 1;
  }
  std::cout << "Wrote " << output_path << "\n";
  std::cout << "GPU processing time (ms): " << gpu_ms << "\n";

  // append to CSV log
  std::ofstream csv;
  csv.open(log_path, std::ios::app);
  if (csv.tellp() == 0) {
    csv << "input,output,width,height,block_size,gpu_ms\n";
  }
  csv << input_path << "," << output_path << "," << w << "," << h << "," << block_size << "," << gpu_ms << "\n";
  csv.close();
  std::cout << "Appended timing to " << log_path << "\n";

  free(out_buf);
  stbi_image_free(img);
  return 0;
}
