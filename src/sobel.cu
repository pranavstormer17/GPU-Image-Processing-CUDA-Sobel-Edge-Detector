#include "hip/hip_runtime.h"
#include "sobel.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>

static void check_cuda(hipError_t e, const char* file, int line) {
  if (e != hipSuccess) {
    std::cerr << "CUDA error " << hipGetErrorString(e)
              << " at " << file << ":" << line << std::endl;
    std::exit(EXIT_FAILURE);
  }
}
#define CHECK_CUDA(x) check_cuda((x), __FILE__, __LINE__)

// Convert RGBA uchar4 into grayscale float
__device__ inline float rgba_to_gray(const uchar4& px) {
  // Rec. 601 luma
  return 0.299f * px.x + 0.587f * px.y + 0.114f * px.z;
}

// Kernel: convert RGBA to grayscale floats in a tile using shared mem,
// then compute Sobel and write result back as RGBA (edge intensity on all channels).
template<int BLOCK>
__global__ void sobel_kernel(const uchar4* input,
                             uint8_t* output,
                             int width, int height, int pitch_in_pixels) {
  // tile includes halo of 1 pixel for Sobel
  __shared__ float tile[BLOCK + 2][BLOCK + 2];

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * BLOCK + tx;
  int y = blockIdx.y * BLOCK + ty;

  // Load into shared memory including halo
  int sx = tx + 1;
  int sy = ty + 1;

  float val = 0.0f;
  if (x < width && y < height) {
    uchar4 px = input[y * pitch_in_pixels + x];
    val = rgba_to_gray(px);
  }
  tile[sy][sx] = val;

  // load halo (left/right/top/bottom)
  if (tx == 0) {
    int lx = x - 1;
    if (lx >= 0 && y < height) {
      uchar4 px = input[y * pitch_in_pixels + lx];
      tile[sy][0] = rgba_to_gray(px);
    } else {
      tile[sy][0] = 0.0f;
    }
  }
  if (tx == BLOCK - 1) {
    int rx = x + 1;
    if (rx < width && y < height) {
      uchar4 px = input[y * pitch_in_pixels + rx];
      tile[sy][BLOCK + 1] = rgba_to_gray(px);
    } else {
      tile[sy][BLOCK + 1] = 0.0f;
    }
  }
  if (ty == 0) {
    int ty_y = y - 1;
    if (ty_y >= 0 && x < width) {
      uchar4 px = input[ty_y * pitch_in_pixels + x];
      tile[0][sx] = rgba_to_gray(px);
    } else {
      tile[0][sx] = 0.0f;
    }
  }
  if (ty == BLOCK - 1) {
    int by = y + 1;
    if (by < height && x < width) {
      uchar4 px = input[by * pitch_in_pixels + x];
      tile[BLOCK + 1][sx] = rgba_to_gray(px);
    } else {
      tile[BLOCK + 1][sx] = 0.0f;
    }
  }

  // corners
  if (tx == 0 && ty == 0) {
    int cx = x - 1;
    int cy = y - 1;
    if (cx >= 0 && cy >= 0) {
      uchar4 px = input[cy * pitch_in_pixels + cx];
      tile[0][0] = rgba_to_gray(px);
    } else {
      tile[0][0] = 0.0f;
    }
  }
  if (tx == BLOCK - 1 && ty == 0) {
    int cx = x + 1;
    int cy = y - 1;
    if (cx < width && cy >= 0) {
      uchar4 px = input[cy * pitch_in_pixels + cx];
      tile[0][BLOCK + 1] = rgba_to_gray(px);
    } else {
      tile[0][BLOCK + 1] = 0.0f;
    }
  }
  if (tx == 0 && ty == BLOCK - 1) {
    int cx = x - 1;
    int cy = y + 1;
    if (cx >= 0 && cy < height) {
      uchar4 px = input[cy * pitch_in_pixels + cx];
      tile[BLOCK + 1][0] = rgba_to_gray(px);
    } else {
      tile[BLOCK + 1][0] = 0.0f;
    }
  }
  if (tx == BLOCK - 1 && ty == BLOCK - 1) {
    int cx = x + 1;
    int cy = y + 1;
    if (cx < width && cy < height) {
      uchar4 px = input[cy * pitch_in_pixels + cx];
      tile[BLOCK + 1][BLOCK + 1] = rgba_to_gray(px);
    } else {
      tile[BLOCK + 1][BLOCK + 1] = 0.0f;
    }
  }

  __syncthreads();

  if (x >= width || y >= height) return;

  // Sobel filters
  float gx = -tile[sy-1][sx-1] - 2.0f * tile[sy][sx-1] - tile[sy+1][sx-1]
             + tile[sy-1][sx+1] + 2.0f * tile[sy][sx+1] + tile[sy+1][sx+1];
  float gy = -tile[sy-1][sx-1] - 2.0f * tile[sy-1][sx] - tile[sy-1][sx+1]
             + tile[sy+1][sx-1] + 2.0f * tile[sy+1][sx] + tile[sy+1][sx+1];

  float mag = sqrtf(gx*gx + gy*gy);
  // normalize to [0,255]
  float outv = fminf(255.0f, mag);

  // write RGBA output (keep alpha = 255)
  int out_idx = (y * width + x) * 4;
  output[out_idx + 0] = static_cast<uint8_t>(outv);
  output[out_idx + 1] = static_cast<uint8_t>(outv);
  output[out_idx + 2] = static_cast<uint8_t>(outv);
  output[out_idx + 3] = 255;
}

void sobel_process_image(const uint8_t* input_rgba,
                         uint8_t* output_rgba,
                         int width,
                         int height,
                         int input_stride,
                         int output_stride,
                         int block_size,
                         float* gpu_elapsed_ms) {
  // input_rgba is host RGBA bytes (4*width*height),
  // pitch in pixels for device pointer we will use equals width.
  size_t num_pixels = static_cast<size_t>(width) * height;
  size_t buffer_bytes = num_pixels * 4;

  uchar4* d_input = nullptr;
  uint8_t* d_output = nullptr;

  CHECK_CUDA(hipMalloc(&d_input, num_pixels * sizeof(uchar4)));
  CHECK_CUDA(hipMalloc(&d_output, buffer_bytes));

  // copy host rgba into d_input (uchar4 layout)
  CHECK_CUDA(hipMemcpy(d_input, input_rgba, buffer_bytes, hipMemcpyHostToDevice));

  dim3 block(block_size, block_size);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

  // timing with CUDA events
  hipEvent_t start, stop;
  CHECK_CUDA(hipEventCreate(&start));
  CHECK_CUDA(hipEventCreate(&stop));

  CHECK_CUDA(hipEventRecord(start));
  // Launch kernel with template block size
  if (block_size == 8) {
    sobel_kernel<8><<<grid, block>>> (d_input, d_output, width, height, width);
  } else if (block_size == 16) {
    sobel_kernel<16><<<grid, block>>> (d_input, d_output, width, height, width);
  } else if (block_size == 32) {
    sobel_kernel<32><<<grid, block>>> (d_input, d_output, width, height, width);
  } else {
    // fallback to 16
    sobel_kernel<16><<<grid, dim3(16,16)>>> (d_input, d_output, width, height, width);
  }
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipEventRecord(stop));
  CHECK_CUDA(hipEventSynchronize(stop));

  float ms = 0.0f;
  CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));
  if (gpu_elapsed_ms) *gpu_elapsed_ms = ms;

  // copy back
  CHECK_CUDA(hipMemcpy(output_rgba, d_output, buffer_bytes, hipMemcpyDeviceToHost));

  // cleanup
  CHECK_CUDA(hipEventDestroy(start));
  CHECK_CUDA(hipEventDestroy(stop));
  CHECK_CUDA(hipFree(d_input));
  CHECK_CUDA(hipFree(d_output));
}
